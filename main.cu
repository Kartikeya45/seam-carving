#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "./src/library.h"

using namespace std;
//

int WIDTH;
__device__ int d_WIDTH;

int xSobel[3][3] = {{1,0,-1},{2,0,-2},{1,0,-1}};
int ySobel[3][3] = {{1,2,1},{0,0,0},{-1,-2,-1}};
__constant__ int d_xSobel[9] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
__constant__ int d_ySobel[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
const int filterWidth = 3;


/**
 * @param argc[1] name of the input file (.pmn)
 * @param argc[2] name of output file with no extension, created by using host & device
 * @param argc[3] horizontal of image you want to resize 
 * @param argc[4] - optional - default(32): blocksize.x
 * @param argc[5] - optional - default(32): blocksize.y
 */
void checkInput(int argc, char ** argv, int &width, int &height, uchar3 * &rgbPic, int &desiredWidth, dim3 &blockSize) {
    if (argc != 4 && argc != 6) {
        printf("The number of arguments is invalid\n");
        exit(EXIT_FAILURE);
    }

    // Read file
    readPnm(argv[1], width, height, rgbPic);
    printf("Image size (width x height): %i x %i\n\n", width, height);

    WIDTH = width;
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_WIDTH), &width, sizeof(int)));

    // Check user's desired width
    desiredWidth = atoi(argv[3]);

    if (desiredWidth <= 0 || desiredWidth >= width) {
        printf("Your desired width must between 0 & current picture's width!\n");
        exit(EXIT_FAILURE);
    }

    // Block size
    if (argc == 6) {
        blockSize.x = atoi(argv[4]);
        blockSize.y = atoi(argv[5]);
    } 

    // Check GPU is working or not
    printDeviceInfo();
}


__global__ void convertRgb2GrayKernel(uchar3 * rgbPic, int width, int height, uint8_t * grayPic) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width) {
        int i = r * width + c;
        grayPic[i] = 0.299f*rgbPic[i].x + 0.587f*rgbPic[i].y + 0.114f*rgbPic[i].z;
    }
}


__global__ void calEnergy(uint8_t * inPixels, int width, int height, int * energy) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int s_width = blockDim.x + filterWidth - 1;
    int s_height = blockDim.y + filterWidth - 1;

    // Each block loads data from GMEM to SMEM
    extern __shared__ uint8_t s_inPixels[];

    int readRow = row - filterWidth / 2, readCol, tmpRow, tmpCol;
    int firstReadCol = col - filterWidth / 2;
    int virtualRow, virtualCol;

    for (virtualRow = threadIdx.y; virtualRow < s_height; readRow += blockDim.y, virtualRow += blockDim.y) {
        tmpRow = readRow;

        // if (readRow < 0)
        //     readRow = 0;
        // else if (readRow >= height) 
        //     readRow = height - 1;

        readRow = min(max(readRow, 0), height - 1);//0 <= readCol <= height-1
        
        readCol = firstReadCol;
        virtualCol = threadIdx.x;

        for (; virtualCol < s_width; readCol += blockDim.x, virtualCol += blockDim.x) {
            tmpCol = readCol;

            // if (readCol < 0) 
            //     readCol = 0;
            // else if (readCol >= width) 
            //     readCol = width - 1;

            readCol = min(max(readCol, 0), width - 1);// 0 <= readCol <= width-1
            
            s_inPixels[virtualRow * s_width + virtualCol] = inPixels[readRow * d_WIDTH + readCol];
            readCol = tmpCol;
        }
        readRow = tmpRow;
    } 
    __syncthreads();


    // Each thread compute energy on SMEM
    int x_kernel = 0, y_kernel = 0;
    for (int i = 0; i < filterWidth; ++i) {
        for (int j = 0; j < filterWidth; ++j) {
            uint8_t closest = s_inPixels[(threadIdx.y + i) * s_width + threadIdx.x + j];
            int filterIdx = i * filterWidth + j;
            x_kernel += closest * d_xSobel[filterIdx];
            y_kernel += closest * d_ySobel[filterIdx];
        }
    }

    // Each thread writes result from SMEM to GMEM
    if (col < width && row < height)
        energy[row * d_WIDTH + col] = abs(x_kernel) + abs(y_kernel);
}

__global__ void carvingKernel(int * leastSignificantPixel, uchar3 * outPixels, uint8_t *grayPixels, int * energy, int width) {
    int row = blockIdx.x;
    int baseIdx = row * d_WIDTH;
    for (int i = leastSignificantPixel[row]; i < width - 1; ++i) {
        outPixels[baseIdx + i] = outPixels[baseIdx + i + 1];
        grayPixels[baseIdx + i] = grayPixels[baseIdx + i + 1];
        energy[baseIdx + i] = energy[baseIdx + i + 1];
    }
}

void findSeam(int * minimalEnergy, int *leastSignificantPixel, int width, int height) {
    int minCol = 0, r = height - 1;

    for (int c = 1; c < width; ++c)
        if (minimalEnergy[r * WIDTH + c] < minimalEnergy[r * WIDTH + minCol])
            minCol = c;
    
    for (; r >= 0; --r) {
        leastSignificantPixel[r] = minCol;
        if (r > 0) {
            int aboveIdx = (r - 1) * WIDTH + minCol;
            int min = minimalEnergy[aboveIdx], minColCpy = minCol;

            if (minColCpy > 0 && minimalEnergy[aboveIdx - 1] < min) {
                min = minimalEnergy[aboveIdx - 1];
                minCol = minColCpy - 1;
            }
            if (minColCpy < width - 1 && minimalEnergy[aboveIdx + 1] < min) {
                minCol = minColCpy + 1;
            }
        }
    }
}

__global__ void energyToTheEndKernel(int * energy, int * minimalEnergy, int width, int height, int fromRow) {
    size_t halfBlock = blockDim.x / 2;//blockDim.x >> 1

    int col = blockIdx.x * halfBlock - halfBlock + threadIdx.x;

    if (fromRow == 0 && col >= 0 && col < width) {
        minimalEnergy[col] = energy[col];
    }
    __syncthreads();

    for (int stride = fromRow != 0 ? 0 : 1; stride < halfBlock && fromRow + stride < height; ++stride) {
        if (threadIdx.x < blockDim.x - (stride << 1)) {
            int curRow = fromRow + stride;
            int curCol = col + stride;

            if (curCol >= 0 && curCol < width) {
                int idx = curRow * d_WIDTH + curCol;
                int aboveIdx = (curRow - 1) * d_WIDTH + curCol;

                int min = minimalEnergy[aboveIdx];
                if (curCol > 0 && minimalEnergy[aboveIdx - 1] < min)
                    min = minimalEnergy[aboveIdx - 1];
                
                if (curCol < width - 1 && minimalEnergy[aboveIdx + 1] < min)
                    min = minimalEnergy[aboveIdx + 1];
                

                minimalEnergy[idx] = min + energy[idx];
            }
        }
        __syncthreads();
    }
}

void deviceResizing(uchar3 * inPixels, int width, int height, int desiredWidth, uchar3 * outPixels, dim3 blockSize) {
    GpuTimer timer;
    timer.Start();

    // allocate kernel memory
    uchar3 * d_inPixels;
    CHECK(hipMalloc(&d_inPixels, width * height * sizeof(uchar3)));
    uint8_t * d_grayPixels;
    CHECK(hipMalloc(&d_grayPixels, width * height * sizeof(uint8_t)));
    int * d_energy;
    CHECK(hipMalloc(&d_energy, width * height * sizeof(int)));
    int * d_leastSignificantPixel;
    CHECK(hipMalloc(&d_leastSignificantPixel, height * sizeof(int)));
    int * d_minimalEnergy;
    CHECK(hipMalloc(&d_minimalEnergy, width * height * sizeof(int)));

    // allocate host memory
    int * energy = (int *)malloc(width * height * sizeof(int));
    int * leastSignificantPixel = (int *)malloc(height * sizeof(int));
    int * minimalEnergy = (int *)malloc(width * height * sizeof(int));

    // dynamically sized smem used to compute energy
    size_t smemSize = ((blockSize.x + 3 - 1) * (blockSize.y + 3 - 1)) * sizeof(uint8_t);
    
    // block size use to calculate minimal energy to the end
    int blockSizeDp = 256;
    int gridSizeDp = (((width - 1) / blockSizeDp + 1) << 1) + 1;
    int stripHeight = (blockSizeDp >> 1) + 1;

    // copy input to device
    CHECK(hipMemcpy(d_inPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));

    // turn input image to grayscale
    dim3 gridSize((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
    convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_grayPixels);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    while (width > desiredWidth) {
        // update energy
        calEnergy<<<gridSize, blockSize, smemSize>>>(d_grayPixels, width, height, d_energy);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        // compute min seam table
        for (int i = 0; i < height; i += (stripHeight >> 1)) {
            energyToTheEndKernel<<<gridSizeDp, blockSizeDp>>>(d_energy, d_minimalEnergy, width, height, i);
            hipDeviceSynchronize();
            CHECK(hipGetLastError());
        }

        // find least significant pixel index of each row and store in d_leastSignificantPixel (SEQUENTIAL, in kernel or host)
        CHECK(hipMemcpy(minimalEnergy, d_minimalEnergy, WIDTH * height * sizeof(int), hipMemcpyDeviceToHost));
        findSeam(minimalEnergy, leastSignificantPixel, width, height);

        // carve
        CHECK(hipMemcpy(d_leastSignificantPixel, leastSignificantPixel, height * sizeof(int), hipMemcpyHostToDevice));
        carvingKernel<<<height, 1>>>(d_leastSignificantPixel, d_inPixels, d_grayPixels, d_energy, width);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        
        --width;
    }

    CHECK(hipMemcpy(outPixels, d_inPixels, WIDTH * height * sizeof(uchar3), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_grayPixels));
    CHECK(hipFree(d_energy));
    CHECK(hipFree(d_leastSignificantPixel));
    CHECK(hipFree(d_minimalEnergy));

    free(minimalEnergy);
    free(leastSignificantPixel);
    free(energy);

    timer.Stop();
    timer.printTime((char *)"device");   
}

int main(int argc, char ** argv) {   

    int width, height, desiredWidth;
    uchar3 * rgbPic;
    dim3 blockSize(32, 32);

    // Check user's input
    checkInput(argc, argv, width, height, rgbPic, desiredWidth, blockSize);

    // DEVICE
    uchar3 * out_device = (uchar3 *)malloc(width * height * sizeof(uchar3));
    deviceResizing(rgbPic, width, height, desiredWidth, out_device, blockSize);

    // Compute error
    // printError((char * )"Error between device result and host result: ", out_host, out_device, width, height);

    // Write result to file, to pnm format
    writePnm(out_device, desiredWidth, height, width, concatStr(argv[2], "_device.pnm"));

    // Free memories
    free(rgbPic);
    free(out_device);
}
