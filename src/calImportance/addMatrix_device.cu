
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

__global__ void addMatrix_kernel(uint8_t *in1, uint8_t *in2, int nRows, int nCols, uint8_t *out) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < nRows && c < nCols) { 
        int i = r * nCols + c;
        out[i] = in1[i] + in2[i];
    }
}