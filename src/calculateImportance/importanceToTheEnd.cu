#include <stdio.h>
#include <stdint.h>
#include "../library.h"


/**
 * @param useDevice = false (default)
 * @param blockSize = dim3(32,32) (default)
 */
void importanceToTheEnd(uint8_t * inPixels, int height, int width, uint8_t * outPixels, bool useDevice, dim3 blockSize) {
    GpuTimer timer;
	timer.Start();

    if (useDevice == false) 
        outPixels = importanceToTheEnd_host(inPixels, width, height, outPixels);
	else { 
        // Allocate device memories
        uint8_t * d_inPixels, * d_outPixels; // d_outPixels is empty. No need to hipMemcpy
        size_t nBytes = height * width * sizeof(uint8_t);
        CHECK(hipMalloc(&d_inPixels, nBytes));
        CHECK(hipMalloc(&d_outPixels, nBytes));

        // Copy data to device memories
        CHECK(hipMemcpy(d_inPixels, inPixels, nBytes, hipMemcpyHostToDevice));

        // Set grid size and call kernel
        dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
        // importanceToTheEnd_device<<<gridSize, blockSize>>>(d_inPixels, width, height, d_outPixels);

        // Copy result from device memory
        //CHECK(hipMemcpy(outPixels, d_outPixels, nBytes, hipMemcpyDeviceToHost));

        // Free device memories
        CHECK(hipFree(d_inPixels));
        CHECK(hipFree(d_outPixels));
    }

    timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n\n", useDevice == true? "use device" : "use host", time);
}