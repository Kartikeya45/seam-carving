#include <stdio.h>
#include "../library.h"

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

/**
 * gray = 0.299*red + 0.587*green + 0.114*blue  
 */
void convertRgb2Gray(uint8_t * inPixels, int width, int height, uint8_t * outPixels, bool useDevice, dim3 blockSize) {
	GpuTimer timer;//
	timer.Start();//
	if (useDevice == false)
    convertRgb2Gray_host(inPixels, width, height, outPixels);
	
	else { // use device
		// Allocate device memories
    uint8_t * d_inPixels;
    uint8_t * d_outPixels;
    size_t nBytes = width * height * sizeof(int);
    CHECK(hipMalloc(&d_inPixels, nBytes));
    CHECK(hipMalloc(&d_outPixels, nBytes));

		// Copy data to device memories
    CHECK(hipMemcpy(d_inPixels, inPixels, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_outPixels, outPixels, nBytes, hipMemcpyHostToDevice));

		// Set grid size and call kernel
    dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
    convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_outPixels);

    // Check kernel error
		hipError_t errSync  = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess) 
		  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
		  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

		// Copy result from device memories
    CHECK(hipMemcpy(outPixels, d_outPixels, nBytes, hipMemcpyDeviceToHost));

		// Free device memories
    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_outPixels));
	}
	timer.Stop();//
	float time = timer.Elapsed();//
	printf("Processing time (%s): %f ms\n\n", useDevice == true? "use device" : "use host", time);
}