#include "hip/hip_runtime.h"
__global__ void convertRgb2GrayKernel(uint8_t * inPixels, int width, int height, 
		uint8_t * outPixels)
{
	// TODO
    // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue  
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width)
    { 
		int i = r * width + c;
		uint8_t red = inPixels[3 * i];
		uint8_t green = inPixels[3 * i + 1];
		uint8_t blue = inPixels[3 * i + 2];
		outPixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
    }
}

void convertRgb2Gray(uint8_t * inPixels, int width, int height,
		uint8_t * outPixels, 
		bool useDevice=false, dim3 blockSize=dim3(1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
        // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue  
        for (int r = 0; r < height; r++)
        {
            for (int c = 0; c < width; c++)
            {
                int i = r * width + c;
                uint8_t red = inPixels[3 * i];
                uint8_t green = inPixels[3 * i + 1];
                uint8_t blue = inPixels[3 * i + 2];
                outPixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
            }
        }
	}
	else // use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO: Allocate device memories
        uint8_t * d_inPixels;
        uint8_t * d_outPixels;
        size_t nBytes = width * height * sizeof(int);
        CHECK(hipMalloc(&d_inPixels, nBytes));
        CHECK(hipMalloc(&d_outPixels, nBytes));

		// TODO: Copy data to device memories
        CHECK(hipMemcpy(d_inPixels, inPixels, nBytes, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_outPixels, outPixels, nBytes, hipMemcpyHostToDevice));

		// TODO: Set grid size and call kernel (remember to check kernel error)
        dim3 gridSize((width - 1) / blockSize.x + 1,
                (height - 1) / blockSize.y + 1);
        convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_outPixels);

		hipError_t errSync  = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess) 
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

		// TODO: Copy result from device memories
        CHECK(hipMemcpy(outPixels, d_outPixels, nBytes, hipMemcpyDeviceToHost));

		// TODO: Free device memories
        CHECK(hipFree(d_inPixels));
        CHECK(hipFree(d_outPixels));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n\n", 
			useDevice == true? "use device" : "use host", time);
}