#include <stdio.h>
#include <stdint.h>
#include "library.h"

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

void step0_checkInput(int argc, char ** argv, dim3 &blockSize) {
    if (argc != 4 && argc != 6) {
		printf("The number of arguments is invalid\n");
		exit(EXIT_FAILURE);
	}

    int width, height;
    uint8_t * inPixels;

    readPnm(argv[1], width, height, inPixels);
	
    // Check if we can resize the width to be smaller or not
    int newWidthSize = atoi(argv[3]);
    if (newWidthSize >= width || newWidthSize <= 0) {
        printf("The size you've given is wrong!\n");
		exit(EXIT_FAILURE);
    }

    // Get block Size
    if (argc == 5) {
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	}

    // Check if GPU is still working
    printDeviceInfo();
}


void step1_convertRgb2Gray(char * inFileName, char * outFileName, dim3 blockSize, int version) {
    int width, height;
    uint8_t * inPixels;

    readPnm(inFileName, width, height, inPixels);

    // host
    uint8_t * correctOutPixels = (uint8_t *)malloc(width * height);
	convertRgb2Gray(inPixels, width, height, correctOutPixels);

    //device (version 1)
    uint8_t * outPixels = (uint8_t *)malloc(width * height);
    convertRgb2Gray(inPixels, width, height, outPixels, true, blockSize);
	
    printError("RGB TO GRAYSCALE - Error ", outPixels, correctOutPixels, width, height);

    // Write the Grayscale images 
	writePnm(correctOutPixels, width, height, outFileName);
	writePnm(outPixels, width, height, outFileName);

    //free memory
    free(inPixels);
    free(correctOutPixels);
    free(outPixels);
}


void step2_detectEdges(char * inFileName, char * outFileName, dim3 blockSize, int version, char * filterName) {
    // Read x-sobel & y-sobel
    int filterWidth;
    float * filter;

    if (strcmp(filterName, "x")) {
        filter = readFilter("../filter/x-sobel.txt", filterWidth);

    } else if (strcmp(filterName, "y")) {
        filter = readFilter("../filter/y-sobel.txt", filterWidth);
    }
    
    // if x-sobel & y-sobel doesn't have the same width or their width is even:
    if (filterWidth % 2 == 0) {
        printf("Filters are not suitable!\n");
		exit(EXIT_FAILURE);
    }

    //Read grayscale image
    int width, height;
    uint8_t * inPixels;
    readPnm(inFileName, width, height, inPixels);

    // apply using host
    uint8_t * outPixels_host = (uint8_t *)malloc(width * height * sizeof(uint8_t)); 
	convolution(inPixels, width, height, filter, filterWidth, outPixels_host);
    //writePnm(outPixels_host, width, height, outFileName);

    uint8_t * outPixels_device = (uint8_t *)malloc(width * height * sizeof(uint8_t));

    if (version == 1) {
        // Apply using device (version 1)
        convolution(inPixels, width, height, filter, filterWidth, outPixels_device, true, blockSize, 1);
        printError("CONVOLUTION - Error (version 1)", outPixels_device, outPixels_host, width, height);
    }
    else if (version == 2) {
        // Apply using device (version 2)
        convolution(inPixels, width, height, filter, filterWidth, outPixels_device, true, blockSize, 2);
        printError("CONVOLUTION - Error (version 2)", outPixels_device, outPixels_host, width, height);
    }
	else if (version == 3) {
        // Apply using device (version 3)
        convolution(inPixels, width, height, filter, filterWidth, outPixels_device, true, blockSize, 3);
        printError("CONVOLUTION - Error (version 3)", outPixels_device, outPixels_host, width, height);
    }
    // write file
    writePnm(outPixels_device, width, height, outFileName);

    // free memory
    free(inPixels);
    free(outPixels_host);
    free(outPixels_device);
}


/**
 * @param argc[1] name of the input file (.pmn)
 * @param argc[2] name of output file with no extension, created by using host & device
 * @param argc[3] horizontal of image you want to resize 
 * @param argc[4] - optional - default(32): blocksize.x
 * @param argc[5] - optional - default(32): blocksize.y
 * @see HW1_P2
 */
int main(int argc, char ** argv) {

    // STEP 0
    dim3 blockSize(32, 32);
    step0_checkInput(argc, argv, blockSize);

    // STEP 1
    char * grayscalePath = concatStr(argv[2], "_grayscale_host.pnm");
    step1_convertRgb2Gray(argv[1], grayscalePath, blockSize, 1);

    // STEP 2
    step2_detectEdges(grayscalePath, concatStr(argv[2], "x_edge_v1.pnm"), blockSize, 1, "x");
    step2_detectEdges(grayscalePath, concatStr(argv[2], "x_edge_v2.pnm"), blockSize, 2, "x");
    step2_detectEdges(grayscalePath, concatStr(argv[2], "x_edge_v3.pnm"), blockSize, 3, "x");

    step2_detectEdges(grayscalePath, concatStr(argv[2], "y_edge_v1.pnm"), blockSize, 1, "x");
    step2_detectEdges(grayscalePath, concatStr(argv[2], "y_edge_v2.pnm"), blockSize, 2, "x");
    step2_detectEdges(grayscalePath, concatStr(argv[2], "y_edge_v3.pnm"), blockSize, 3, "x");

    // STEP 3

    // Calculate the importance from the end

    // Find & Erase seam

    
}