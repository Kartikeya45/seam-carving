#include <stdio.h>
#include "library.h"

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

/**
 * @param argc[1] name of the input file (.pmn)
 * @param argc[2] name of output file with no extension, created by using host & device
 * @param argc[3] horizontal of image you want to resize 
 * @param argc[4] - optional - default(32): blocksize.x
 * @param argc[5] - optional - default(32): blocksize.y
 * @see HW1_P2
 */
int main(int argc, char ** argv) {
    if (argc != 4 && argc != 6) {
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}
	
	printDeviceInfo();
	
  // Read user's args
  dim3 blockSize(32, 32); // Default
  int resize = argv[3];
  if (argc == 5) {
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	}

	// Read input image file
  int numChannels, width, height;
  uint8_t * inPixels;
	readPnm(argv[1], numChannels, width, height, inPixels);

  if (numChannels != 3) {
    printf("Number of channels is not 3, input image must be RGB!");
		return EXIT_FAILURE; 
  }
	printf("Image size (width x height): %i x %i\n\n", width, height);

  // Convert RGB to Grayscale
  uint8_t * correctOutPixels= (uint8_t *)malloc(width * height);
	convertRgb2Gray(inPixels, width, height, correctOutPixels);

  uint8_t * outPixels= (uint8_t *)malloc(width * height);
  convertRgb2Gray(inPixels, width, height, outPixels, true, blockSize);
	
  float err = computeError(outPixels, correctOutPixels, width * height);
	printf("Error after convert RGB to Grayscale: %f\n", err);

  // Write the Grayscale image
	writePnm_uint8_t(correctOutPixels, 1, width, height, concatStr(argv[2], "_host.pnm"));
	writePnm_uint8_t(outPixels, 1, width, height, concatStr(argv[2], "_device.pnm"));

  // Seam carving


	// Write results to files
	//writePnm(inPixels, width, height, concatStr(argv[2], "_host.pnm"));
// 	writePnm(hostOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
// 	writePnm(deviceOutPixels, width, height, concatStr(outFileNameBase, "_device.pnm"));
    
  // Free memories
  free(inPixels);
  free(outPixels);
}